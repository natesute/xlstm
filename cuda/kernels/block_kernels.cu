#include "hip/hip_runtime.h"
// block_kernels.cu

#include <hip/hip_runtime.h>
#include <>
#include <hip/hip_fp16.h>
#include "block_kernels.h"

#include "cuda_utils.h"

// xLSTM block forward pass kernel
template __global__ void xlstm_block_forward_kernel<float>(const float *__restrict__ x,
                                const float *__restrict__ h_prev,
                                const float *__restrict__ c_prev,
                                const float *__restrict__ C_prev,
                                const float *__restrict__ n_prev,
                                float *__restrict__ h,
                                float *__restrict__ c,
                                float *__restrict__ C,
                                float *__restrict__ n,
                                const float *__restrict__ w_proj,
                                const float *__restrict__ w_gate,
                                const float *__restrict__ b_proj,
                                const float *__restrict__ b_gate,
                                const float *__restrict__ w_slstm,
                                const float *__restrict__ w_mlstm,
                                const float *__restrict__ b_slstm,
                                const float *__restrict__ b_mlstm,
                                int batch_size,
                                int input_size,
                                int hidden_size,
                                int proj_size,
                                bool use_mlstm);

// xLSTM block forward pass kernel
template __global__ void xlstm_block_forward_kernel<double>(const double *__restrict__ x,
                                const double *__restrict__ h_prev,
                                const double *__restrict__ c_prev,
                                const double *__restrict__ C_prev,
                                const double *__restrict__ n_prev,
                                double *__restrict__ h,
                                double *__restrict__ c,
                                double *__restrict__ C,
                                double *__restrict__ n,
                                const double *__restrict__ w_proj,
                                const double *__restrict__ w_gate,
                                const double *__restrict__ b_proj,
                                const double *__restrict__ b_gate,
                                const double *__restrict__ w_slstm,
                                const double *__restrict__ w_mlstm,
                                const double *__restrict__ b_slstm,
                                const double *__restrict__ b_mlstm,
                                int batch_size,
                                int input_size,
                                int hidden_size,
                                int proj_size,
                                bool use_mlstm);

template <typename T>
__global__ void slstm_backward_kernel(const T *__restrict__ grad_h,
                                      const T *__restrict__ grad_c,
                                      const T *__restrict__ c,
                                      const T *__restrict__ n,
                                      const T *__restrict__ c_prev,
                                      const T *__restrict__ n_prev,
                                      const T *__restrict__ x,
                                      const T *__restrict__ h_prev,
                                      const T *__restrict__ w_i,
                                      const T *__restrict__ w_f,
                                      const T *__restrict__ w_z,
                                      const T *__restrict__ w_o,
                                      const T *__restrict__ r_i,
                                      const T *__restrict__ r_f,
                                      const T *__restrict__ r_z,
                                      const T *__restrict__ r_o,
                                      const T *__restrict__ b_i,
                                      const T *__restrict__ b_f,
                                      const T *__restrict__ b_z,
                                      const T *__restrict__ b_o,
                                      const T *__restrict__ grad_x,
                                      const T *__restrict__ grad_h_prev,
                                      const T *__restrict__ grad_c_prev,
                                      const T *__restrict__ grad_n_prev,
                                      const T *__restrict__ grad_w_i,
                                      const T *__restrict__ grad_w_f,
                                      const T *__restrict__ grad_w_z,
                                      const T *__restrict__ grad_w_o,
                                      const T *__restrict__ grad_r_i,
                                      const T *__restrict__ grad_r_f,
                                      const T *__restrict__ grad_r_z,
                                      const T *__restrict__ grad_r_o,
                                      const T *__restrict__ grad_b_i,
                                      const T *__restrict__ grad_b_f,
                                      const T *__restrict__ grad_b_z,
                                      const T *__restrict__ grad_b_o,
                                      int batch_size,
                                      int input_size,
                                      int hidden_size);

template __global__ void slstm_backward_kernel<float>(const float *__restrict__ grad_h,
                                      const float *__restrict__ grad_c,
                                      const float *__restrict__ c,
                                      const float *__restrict__ n,
                                      const float *__restrict__ c_prev,
                                      const float *__restrict__ n_prev,
                                      const float *__restrict__ x,
                                      const float *__restrict__ h_prev,
                                      const float *__restrict__ w_i,
                                      const float *__restrict__ w_f,
                                      const float *__restrict__ w_z,
                                      const float *__restrict__ w_o,
                                      const float *__restrict__ r_i,
                                      const float *__restrict__ r_f,
                                      const float *__restrict__ r_z,
                                      const float *__restrict__ r_o,
                                      const float *__restrict__ b_i,
                                      const float *__restrict__ b_f,
                                      const float *__restrict__ b_z,
                                      const float *__restrict__ b_o,
                                      const float *__restrict__ grad_x,
                                      const float *__restrict__ grad_h_prev,
                                      const float *__restrict__ grad_c_prev,
                                      const float *__restrict__ grad_n_prev,
                                      const float *__restrict__ grad_w_i,
                                      const float *__restrict__ grad_w_f,
                                      const float *__restrict__ grad_w_z,
                                      const float *__restrict__ grad_w_o,
                                      const float *__restrict__ grad_r_i,
                                      const float *__restrict__ grad_r_f,
                                      const float *__restrict__ grad_r_z,
                                      const float *__restrict__ grad_r_o,
                                      const float *__restrict__ grad_b_i,
                                      const float *__restrict__ grad_b_f,
                                      const float *__restrict__ grad_b_z,
                                      const float *__restrict__ grad_b_o,
                                      int batch_size,
                                      int input_size,
                                      int hidden_size);

template __global__ void slstm_backward_kernel<double>(const double *__restrict__ grad_h,
                                      const double *__restrict__ grad_c,
                                      const double *__restrict__ c,
                                      const double *__restrict__ n,
                                      const double *__restrict__ c_prev,
                                      const double *__restrict__ n_prev,
                                      const double *__restrict__ x,
                                      const double *__restrict__ h_prev,
                                      const double *__restrict__ w_i,
                                      const double *__restrict__ w_f,
                                      const double *__restrict__ w_z,
                                      const double *__restrict__ w_o,
                                      const double *__restrict__ r_i,
                                      const double *__restrict__ r_f,
                                      const double *__restrict__ r_z,
                                      const double *__restrict__ r_o,
                                      const double *__restrict__ b_i,
                                      const double *__restrict__ b_f,
                                      const double *__restrict__ b_z,
                                      const double *__restrict__ b_o,
                                      const double *__restrict__ grad_x,
                                      const double *__restrict__ grad_h_prev,
                                      const double *__restrict__ grad_c_prev,
                                      const double *__restrict__ grad_n_prev,
                                      const double *__restrict__ grad_w_i,
                                      const double *__restrict__ grad_w_f,
                                      const double *__restrict__ grad_w_z,
                                      const double *__restrict__ grad_w_o,
                                      const double *__restrict__ grad_r_i,
                                      const double *__restrict__ grad_r_f,
                                      const double *__restrict__ grad_r_z,
                                      const double *__restrict__ grad_r_o,
                                      const double *__restrict__ grad_b_i,
                                      const double *__restrict__ grad_b_f,
                                      const double *__restrict__ grad_b_z,
                                      const double *__restrict__ grad_b_o,
                                      int batch_size,
                                      int input_size,
                                      int hidden_size);

template __global__ void slstm_forward_kernel<float>(const float *__restrict__ x,
                                     const float *__restrict__ h_prev,
                                     const float *__restrict__ c_prev,
                                     const float *__restrict__ n_prev,
                                     float *__restrict__ c,
                                     float *__restrict__ n,
                                     float *__restrict__ h,
                                     const float *__restrict__ w_i,
                                     const float *__restrict__ w_f,
                                     const float *__restrict__ w_z,
                                     const float *__restrict__ w_o,
                                     const float *__restrict__ r_i,
                                     const float *__restrict__ r_f,
                                     const float *__restrict__ r_z,
                                     const float *__restrict__ r_o,
                                     const float *__restrict__ b_i,
                                     const float *__restrict__ b_f,
                                     const float *__restrict__ b_z,
                                     const float *__restrict__ b_o,
                                     int batch_size,
                                     int input_size,
                                     int hidden_size);

template __global__ void slstm_forward_kernel<double>(const double *__restrict__ x,
                                     const double *__restrict__ h_prev,
                                     const double *__restrict__ c_prev,
                                     const double *__restrict__ n_prev,
                                     double *__restrict__ c,
                                     double *__restrict__ n,
                                     double *__restrict__ h,
                                     const double *__restrict__ w_i,
                                     const double *__restrict__ w_f,
                                     const double *__restrict__ w_z,
                                     const double *__restrict__ w_o,
                                     const double *__restrict__ r_i,
                                     const double *__restrict__ r_f,
                                     const double *__restrict__ r_z,
                                     const double *__restrict__ r_o,
                                     const double *__restrict__ b_i,
                                     const double *__restrict__ b_f,
                                     const double *__restrict__ b_z,
                                     const double *__restrict__ b_o,
                                     int batch_size,
                                     int input_size,
                                     int hidden_size);

template __global__ void mlstm_backward_kernel(const float *__restrict__ grad_h,
                            const float *__restrict__ C,
                            const float *__restrict__ n,
                            const float *__restrict__ x,
                            const float *__restrict__ w_k,
                            const float *__restrict__ w_v,
                            const float *__restrict__ w_q,
                            const float *__restrict__ w_i,
                            const float *__restrict__ w_f,
                            const float *__restrict__ w_o,
                            const float *__restrict__ b_k,
                            const float *__restrict__ b_v,
                            const float *__restrict__ b_q,
                            const float *__restrict__ b_i,
                            const float *__restrict__ b_f,
                            const float *__restrict__ b_o,
                            const float *__restrict__ grad_x,
                            const float *__restrict__ grad_C_prev,
                            const float *__restrict__ grad_n_prev,
                            const float *__restrict__ grad_w_k,
                            const float *__restrict__ grad_w_v,
                            const float *__restrict__ grad_w_q,
                            const float *__restrict__ grad_w_i,
                            const float *__restrict__ grad_w_f,
                            const float *__restrict__ grad_w_o,
                            const float *__restrict__ grad_b_k,
                            const float *__restrict__ grad_b_v,
                            const float *__restrict__ grad_b_q,
                            const float *__restrict__ grad_b_i,
                            const float *__restrict__ grad_b_f,
                            const float *__restrict__ grad_b_o,
                            int batch_size,
                            int input_size,
                            int hidden_size);

template __global__ void mlstm_backward_kernel(const double *__restrict__ grad_h,
                            const double *__restrict__ C,
                            const double *__restrict__ n,
                            const double *__restrict__ x,
                            const double *__restrict__ w_k,
                            const double *__restrict__ w_v,
                            const double *__restrict__ w_q,
                            const double *__restrict__ w_i,
                            const double *__restrict__ w_f,
                            const double *__restrict__ w_o,
                            const double *__restrict__ b_k,
                            const double *__restrict__ b_v,
                            const double *__restrict__ b_q,
                            const double *__restrict__ b_i,
                            const double *__restrict__ b_f,
                            const double *__restrict__ b_o,
                            const double *__restrict__ grad_x,
                            const double *__restrict__ grad_C_prev,
                            const double *__restrict__ grad_n_prev,
                            const double *__restrict__ grad_w_k,
                            const double *__restrict__ grad_w_v,
                            const double *__restrict__ grad_w_q,
                            const double *__restrict__ grad_w_i,
                            const double *__restrict__ grad_w_f,
                            const double *__restrict__ grad_w_o,
                            const double *__restrict__ grad_b_k,
                            const double *__restrict__ grad_b_v,
                            const double *__restrict__ grad_b_q,
                            const double *__restrict__ grad_b_i,
                            const double *__restrict__ grad_b_f,
                            const double *__restrict__ grad_b_o,
                            int batch_size,
                            int input_size,
                            int hidden_size);

template __global__ void mlstm_forward_kernel(const float *__restrict__ x,
                          const float *__restrict__ h_prev,
                          const float *__restrict__ C_prev,
                          const float *__restrict__ n_prev,
                          float *__restrict__ C,
                          float *__restrict__ n,
                          float *__restrict__ h,
                          const float *__restrict__ w_k,
                          const float *__restrict__ w_v,
                          const float *__restrict__ w_q,
                          const float *__restrict__ w_i,
                          const float *__restrict__ w_f,
                          const float *__restrict__ w_o,
                          const float *__restrict__ b_k,
                          const float *__restrict__ b_v,
                          const float *__restrict__ b_q,
                          const float *__restrict__ b_i,
                          const float *__restrict__ b_f,
                          const float *__restrict__ b_o,
                          int batch_size,
                          int input_size,
                          int hidden_size);

template __global__ void mlstm_forward_kernel(const double *__restrict__ x,
                          const double *__restrict__ h_prev,
                          const double *__restrict__ C_prev,
                          const double *__restrict__ n_prev,
                          double *__restrict__ C,
                          double *__restrict__ n,
                          double *__restrict__ h,
                          const double *__restrict__ w_k,
                          const double *__restrict__ w_v,
                          const double *__restrict__ w_q,
                          const double *__restrict__ w_i,
                          const double *__restrict__ w_f,
                          const double *__restrict__ w_o,
                          const double *__restrict__ b_k,
                          const double *__restrict__ b_v,
                          const double *__restrict__ b_q,
                          const double *__restrict__ b_i,
                          const double *__restrict__ b_f,
                          const double *__restrict__ b_o,
                          int batch_size,
                          int input_size,
                          int hidden_size);

// xLSTM block forward pass kernel
template <typename T>
__global__ void xlstm_block_forward_kernel(const T *__restrict__ x,
                                           const T *__restrict__ h_prev,
                                           const T *__restrict__ c_prev,
                                           const T *__restrict__ C_prev,
                                           const T *__restrict__ n_prev,
                                           T *__restrict__ h,
                                           T *__restrict__ c,
                                           T *__restrict__ C,
                                           T *__restrict__ n,
                                           const T *__restrict__ w_proj,
                                           const T *__restrict__ w_gate,
                                           const T *__restrict__ b_proj,
                                           const T *__restrict__ b_gate,
                                           const T *__restrict__ w_slstm,
                                           const T *__restrict__ w_mlstm,
                                           const T *__restrict__ b_slstm,
                                           const T *__restrict__ b_mlstm,
                                           int batch_size,
                                           int input_size,
                                           int hidden_size,
                                           int proj_size,
                                           bool use_mlstm)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for (int i = idx; i < batch_size * hidden_size; i += stride)
    {
        int batch = i / hidden_size;
        int hidx = i % hidden_size;

        // Projection
        T proj = 0;
        for (int j = 0; j < input_size; j++)
        {
            proj += w_proj[hidx * input_size + j] * x[batch * input_size + j];
        }
        proj += b_proj[hidx];

        // Gate
        T gate = sigmoid(w_gate[hidx] * proj + b_gate[hidx]);

        // sLSTM or mLSTM
        if (use_mlstm)
        {
            // mLSTM forward pass
            const T *w_k = w_mlstm + hidx * proj_size * 3;
            const T *w_v = w_k + proj_size;
            const T *w_q = w_v + proj_size;
            const T *w_i = w_mlstm + hidx * proj_size * 6 + proj_size * 3;
            const T *w_f = w_i + proj_size;
            const T *w_o = w_f + proj_size;
            const T *b_k = b_mlstm + hidx * 6;
            const T *b_v = b_k + 1;
            const T *b_q = b_v + 1;
            const T *b_i = b_q + 1;
            const T *b_f = b_i + 1;
            const T *b_o = b_f + 1;

            mlstm_forward_kernel<T><<<1, 1>>>(x + batch * input_size,
                                              h_prev + batch * hidden_size,
                                              C_prev + i * hidden_size,
                                              n_prev + i,
                                              C + i * hidden_size,
                                              n + i,
                                              h + i,
                                              w_k, w_v, w_q,
                                              w_i, w_f, w_o,
                                              b_k, b_v, b_q,
                                              b_i, b_f, b_o,
                                              1, input_size, hidden_size);
        }
        else
        {
            // sLSTM forward pass
            const T *w_i = w_slstm + hidx * proj_size * 4;
            const T *w_f = w_i + proj_size;
            const T *w_z = w_f + proj_size;
            const T *w_o = w_z + proj_size;
            const T *b_i = b_slstm + hidx * 4;
            const T *b_f = b_i + 1;
            const T *b_z = b_f + 1;
            const T *b_o = b_z + 1;

            slstm_forward_kernel<T><<<1, 1>>>(x + batch * input_size,
                                              h_prev + batch * hidden_size,
                                              c_prev + i,
                                              n_prev + i,
                                              c + i,
                                              n + i,
                                              h + i,
                                              w_i, w_f, w_z, w_o,
                                              NULL, NULL, NULL, NULL,
                                              b_i, b_f, b_z, b_o,
                                              1, input_size, hidden_size);
        }

        // Apply gate
        h[i] = gate * h[i] + (1 - gate) * proj;
    }
}

// xLSTM block backward pass kernel
template <typename T>
__global__ void xlstm_block_backward_kernel(const T *__restrict__ grad_h,
                                            const T *__restrict__ h,
                                            const T *__restrict__ c,
                                            const T *__restrict__ C,
                                            const T *__restrict__ n,
                                            const T *__restrict__ x,
                                            const T *__restrict__ w_proj,
                                            const T *__restrict__ w_gate,
                                            const T *__restrict__ b_gate,
                                            const T *__restrict__ w_slstm,
                                            const T *__restrict__ w_mlstm,
                                            T *__restrict__ grad_x,
                                            T *__restrict__ grad_h_prev,
                                            T *__restrict__ grad_c_prev,
                                            T *__restrict__ grad_C_prev,
                                            T *__restrict__ grad_n_prev,
                                            T *__restrict__ grad_w_proj,
                                            T *__restrict__ grad_w_gate,
                                            T *__restrict__ grad_b_proj,
                                            T *__restrict__ grad_b_gate,
                                            const T *__restrict__ grad_w_slstm,
                                            const T *__restrict__ grad_w_mlstm,
                                            const T *__restrict__ grad_b_slstm,
                                            const T *__restrict__ grad_b_mlstm,
                                            int batch_size,
                                            int input_size,
                                            int hidden_size,
                                            int proj_size,
                                            bool use_mlstm)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for (int i = idx; i < batch_size * hidden_size; i += stride)
    {
        int batch = i / hidden_size;
        int hidx = i % hidden_size;

        // Gradient of gate
        T gate = sigmoid(w_gate[hidx] * h[i] + b_gate[hidx]);
        T grad_gate = grad_h[i] * (h[i] - x[batch * input_size + hidx]) * gate * (1 - gate);

        // Gradient of projection
        T grad_proj = grad_h[i] * (1 - gate);

        // sLSTM or mLSTM backward pass
        if (use_mlstm)
        {
            // mLSTM backward pass
            const T *w_k = w_mlstm + hidx * proj_size * 3;
            const T *w_v = w_k + proj_size;
            const T *w_q = w_v + proj_size;
            const T *w_i = w_mlstm + hidx * proj_size * 6 + proj_size * 3;
            const T *w_f = w_i + proj_size;
            const T *w_o = w_f + proj_size;
            const T *grad_w_k = grad_w_mlstm + hidx * proj_size * 6;
            const T *grad_w_v = grad_w_k + proj_size;
            const T *grad_w_q = grad_w_v + proj_size;
            const T *grad_w_i = grad_w_q + proj_size;
            const T *grad_w_f = grad_w_i + proj_size;
            const T *grad_w_o = grad_w_f + proj_size;
            const T *grad_b_k = grad_b_mlstm + hidx * 6;
            const T *grad_b_v = grad_b_k + 1;
            const T *grad_b_q = grad_b_v + 1;
            const T *grad_b_i = grad_b_q + 1;
            const T *grad_b_f = grad_b_i + 1;
            const T *grad_b_o = grad_b_f + 1;

            mlstm_backward_kernel<T><<<1, 1>>>(grad_h + i,
                                               C + i * hidden_size,
                                               n + i,
                                               x + batch * input_size,
                                               w_k, w_v, w_q,
                                               w_i, w_f, w_o,
                                               grad_x + batch * input_size,
                                               grad_C_prev + i * hidden_size,
                                               grad_n_prev + i,
                                               grad_w_k, grad_w_v, grad_w_q,
                                               grad_w_i, grad_w_f, grad_w_o,
                                               grad_b_k, grad_b_v, grad_b_q,
                                               grad_b_i, grad_b_f, grad_b_o,
                                               1, input_size, hidden_size);
        }
        else
        {
            // sLSTM backward pass
            const T *w_i = w_slstm + hidx * proj_size * 4;
            const T *w_f = w_i + proj_size;
            const T *w_z = w_f + proj_size;
            const T *w_o = w_z + proj_size;
            const T *grad_w_i = grad_w_slstm + hidx * proj_size * 4;
            const T *grad_w_f = grad_w_i + proj_size;
            const T *grad_w_z = grad_w_f + proj_size;
            const T *grad_w_o = grad_w_z + proj_size;
            const T *grad_b_i = grad_b_slstm + hidx * 4;
            const T *grad_b_f = grad_b_i + 1;
            const T *grad_b_z = grad_b_f + 1;
            const T *grad_b_o = grad_b_z + 1;

            slstm_backward_kernel<T><<<1, 1>>>(grad_h + i,
                                               grad_gate,
                                               c + i,
                                               n + i,
                                               NULL,
                                               NULL,
                                               x + batch * input_size,
                                               NULL,
                                               w_i, w_f, w_z, w_o,
                                               NULL, NULL, NULL, NULL,
                                               NULL, NULL, NULL, NULL,
                                               grad_x + batch * input_size,
                                               grad_h_prev + batch * hidden_size,
                                               grad_c_prev + i,
                                               grad_n_prev + i,
                                               grad_w_i, grad_w_f, grad_w_z, grad_w_o,
                                               NULL, NULL, NULL, NULL,
                                               grad_b_i, grad_b_f, grad_b_z, grad_b_o,
                                               1, input_size, hidden_size);
        }

        // Gradient of projection weights and bias
        for (int j = 0; j < input_size; j++)
        {
            atomicAdd(&grad_w_proj[hidx * input_size + j], grad_proj * x[batch * input_size + j]);
        }
        atomicAdd(&grad_b_proj[hidx], grad_proj);

        // Gradient of gate weights and bias
        atomicAdd(&grad_w_gate[hidx], grad_gate * h[i]);
        atomicAdd(&grad_b_gate[hidx], grad_gate);
    }
}

// xLSTM block backward pass kernel
template __global__ void xlstm_block_backward_kernel<float>(const float *__restrict__ grad_h,
                                                const float *__restrict__ h,
                                                const float *__restrict__ c,
                                                const float *__restrict__ C,
                                                const float *__restrict__ n,
                                                const float *__restrict__ x,
                                                const float *__restrict__ w_proj,
                                                const float *__restrict__ w_gate,
                                                const float *__restrict__ b_gate,
                                                const float *__restrict__ w_slstm,
                                                const float *__restrict__ w_mlstm,
                                                const float *__restrict__ grad_x,
                                                const float *__restrict__ grad_h_prev,
                                                const float *__restrict__ grad_c_prev,
                                                const float *__restrict__ grad_C_prev,
                                                const float *__restrict__ grad_n_prev,
                                                const float *__restrict__ grad_w_proj,
                                                const float *__restrict__ grad_w_gate,
                                                const float *__restrict__ grad_b_proj,
                                                const float *__restrict__ grad_b_gate,
                                                const float *__restrict__ grad_w_slstm,
                                                const float *__restrict__ grad_w_mlstm,
                                                const float *__restrict__ grad_b_slstm,
                                                const float *__restrict__ grad_b_mlstm,
                                                int batch_size,
                                                int input_size,
                                                int hidden_size,
                                                int proj_size,
                                                bool use_mlstm);

// xLSTM block backward pass kernel
template __global__ void xlstm_block_backward_kernel<double>(const double *__restrict__ grad_h,
                                                const double *__restrict__ h,
                                                const double *__restrict__ c,
                                                const double *__restrict__ C,
                                                const double *__restrict__ n,
                                                const double *__restrict__ x,
                                                const double *__restrict__ w_proj,
                                                const double *__restrict__ w_gate,
                                                const double *__restrict__ b_gate,
                                                const double *__restrict__ w_slstm,
                                                const double *__restrict__ w_mlstm,
                                                const double *__restrict__ grad_x,
                                                const double *__restrict__ grad_h_prev,
                                                const double *__restrict__ grad_c_prev,
                                                const double *__restrict__ grad_C_prev,
                                                const double *__restrict__ grad_n_prev,
                                                const double *__restrict__ grad_w_proj,
                                                const double *__restrict__ grad_w_gate,
                                                const double *__restrict__ grad_b_proj,
                                                const double *__restrict__ grad_b_gate,
                                                const double *__restrict__ grad_w_slstm,
                                                const double *__restrict__ grad_w_mlstm,
                                                const double *__restrict__ grad_b_slstm,
                                                const double *__restrict__ grad_b_mlstm,
                                                int batch_size,
                                                int input_size,
                                                int hidden_size,
                                                int proj_size,
                                                bool use_mlstm);

// Launch the xLSTM block forward pass kernel
template <typename T>
void launch_xlstm_block_forward(const T *x,
                                const T *h_prev,
                                const T *c_prev,
                                const T *C_prev,
                                const T *n_prev,
                                T *h,
                                T *c,
                                T *C,
                                T *n,
                                const T *w_proj,
                                const T *w_gate,
                                const T *b_proj,
                                const T *b_gate,
                                const T *w_slstm,
                                const T *w_mlstm,
                                const T *b_slstm,
                                const T *b_mlstm,
                                int batch_size,
                                int input_size,
                                int hidden_size,
                                int proj_size,
                                bool use_mlstm)
{
    dim3 block(256);
    dim3 grid((batch_size * hidden_size + block.x - 1) / block.x);

    xlstm_block_forward_kernel<T><<<grid, block>>>(x, h_prev, c_prev, C_prev, n_prev,
                                                   h, c, C, n,
                                                   w_proj, w_gate,
                                                   b_proj, b_gate,
                                                   w_slstm, w_mlstm,
                                                   b_slstm, b_mlstm,
                                                   batch_size, input_size, hidden_size, proj_size,
                                                   use_mlstm);
}

// Launch the xLSTM block backward pass kernel
template <typename T>
void launch_xlstm_block_backward(const T *grad_h,
                                 const T *h,
                                 const T *c,
                                 const T *C,
                                 const T *n,
                                 const T *x,
                                 const T *w_proj,
                                 const T *w_gate,
                                 const T *b_gate,
                                 const T *w_slstm,
                                 const T *w_mlstm,
                                 const T *grad_x,
                                 const T *grad_h_prev,
                                 const T *grad_c_prev,
                                 const T *grad_C_prev,
                                 const T *grad_n_prev,
                                 const T *grad_w_proj,
                                 const T *grad_w_gate,
                                 const T *grad_b_proj,
                                 const T *grad_b_gate,
                                 const T *grad_w_slstm,
                                 const T *grad_w_mlstm,
                                 const T *grad_b_slstm,
                                 const T *grad_b_mlstm,
                                 int batch_size,
                                 int input_size,
                                 int hidden_size,
                                 int proj_size,
                                 bool use_mlstm)
{
    dim3 block(256);
    dim3 grid((batch_size * hidden_size + block.x - 1) / block.x);

    xlstm_block_backward_kernel<T><<<grid, block>>>(grad_h,
                                                    h, c, C, n,
                                                    x,
                                                    w_proj, w_gate, b_gate, w_slstm, w_mlstm,
                                                    grad_x,
                                                    grad_h_prev, grad_c_prev,
                                                    grad_C_prev, grad_n_prev,
                                                    grad_w_proj, grad_w_gate,
                                                    grad_b_proj, grad_b_gate,
                                                    grad_w_slstm, grad_w_mlstm,
                                                    grad_b_slstm, grad_b_mlstm,
                                                    batch_size, input_size, hidden_size, proj_size,
                                                    use_mlstm);
}

template void launch_xlstm_block_forward<float>(const float *x,
                                                const float *h_prev,
                                                const float *c_prev,
                                                const float *C_prev,
                                                const float *n_prev,
                                                float *h,
                                                float *c,
                                                float *C,
                                                float *n,
                                                const float *w_proj,
                                                const float *w_gate,
                                                const float *b_proj,
                                                const float *b_gate,
                                                const float *w_slstm,
                                                const float *w_mlstm,
                                                const float *b_slstm,
                                                const float *b_mlstm,
                                                int batch_size,
                                                int input_size,
                                                int hidden_size,
                                                int proj_size,
                                                bool use_mlstm);

template void launch_xlstm_block_forward<double>(const double *x,
                                                const double *h_prev,
                                                const double *c_prev,
                                                const double *C_prev,
                                                const double *n_prev,
                                                double *h,
                                                double *c,
                                                double *C,
                                                double *n,
                                                const double *w_proj,
                                                const double *w_gate,
                                                const double *b_proj,
                                                const double *b_gate,
                                                const double *w_slstm,
                                                const double *w_mlstm,
                                                const double *b_slstm,
                                                const double *b_mlstm,
                                                int batch_size,
                                                int input_size,
                                                int hidden_size,
                                                int proj_size,
                                                bool use_mlstm);

template void launch_xlstm_block_backward<float>(const float *grad_h,
                                                const float *h,
                                                const float *c,
                                                const float *C,
                                                const float *n,
                                                const float *x,
                                                const float *w_proj,
                                                const float *w_gate,
                                                const float *b_gate,
                                                const float *w_slstm,
                                                const float *w_mlstm,
                                                const float *grad_x,
                                                const float *grad_h_prev,
                                                const float *grad_c_prev,
                                                const float *grad_C_prev,
                                                const float *grad_n_prev,
                                                const float *grad_w_proj,
                                                const float *grad_w_gate,
                                                const float *grad_b_proj,
                                                const float *grad_b_gate,
                                                const float *grad_w_slstm,
                                                const float *grad_w_mlstm,
                                                const float *grad_b_slstm,
                                                const float *grad_b_mlstm,
                                                int batch_size,
                                                int input_size,
                                                int hidden_size,
                                                int proj_size,
                                                bool use_mlstm);

template void launch_xlstm_block_backward<double>(const double *grad_h,
                                                const double *h,
                                                const double *c,
                                                const double *C,
                                                const double *n,
                                                const double *x,
                                                const double *w_proj,
                                                const double *w_gate,
                                                const double *b_gate,
                                                const double *w_slstm,
                                                const double *w_mlstm,
                                                const double *grad_x,
                                                const double *grad_h_prev,
                                                const double *grad_c_prev,
                                                const double *grad_C_prev,
                                                const double *grad_n_prev,
                                                const double *grad_w_proj,
                                                const double *grad_w_gate,
                                                const double *grad_b_proj,
                                                const double *grad_b_gate,
                                                const double *grad_w_slstm,
                                                const double *grad_w_mlstm,
                                                const double *grad_b_slstm,
                                                const double *grad_b_mlstm,
                                                int batch_size,
                                                int input_size,
                                                int hidden_size,
                                                int proj_size,
                                                bool use_mlstm);